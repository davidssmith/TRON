#include "hip/hip_runtime.h"
/*
  This file is part of the TRON package (http://github.com/davidssmith/tron).

  The MIT License (MIT)

  Copyright (c) 2016 David Smith

  Permission is hereby granted, free of charge, to any person obtaining a # copy
  of this software and associated documentation files (the "Software"), to # deal
  in the Software without restriction, including without limitation the # rights
  to use, copy, modify, merge, publish, distribute, sublicense, and/or # sell
  copies of the Software, and to permit persons to whom the Software is
  furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included # in all
  copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS # OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL # THE
  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING # FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS # IN THE
  SOFTWARE.
*/

#include <stdio.h>
#include <fcntl.h>
#include <unistd.h>
#include <err.h>
#include <errno.h>
#include <string.h>
#include <math.h>
#include <complex.h>
#include <time.h>
#include <stdint.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include "float2math.h"
#include "mri.h"
#include "ra.h"
#include "tron.h"


// CONFIGURATION PARAMETERS
// TODO: softcode as many as possible
#define NSTREAMS        2
#define MULTI_GPU       0
#define NCHAN           6
#define MAXCHAN         6
#define MAX_RECON_CMDS  20
static const int blocksize = 96;    // TWEAK: CUDA kernel parameters, optimize for your device
static const int gridsize = 2048;
static char flag_verbose = 0;

// GLOBAL VARIABLES
static float2 *d_indata[NSTREAMS], *d_outdata[NSTREAMS], *d_apodos[NSTREAMS], *d_apod[NSTREAMS], *d_tmp[NSTREAMS];
static hipfftHandle fft_plan[NSTREAMS], fft_plan_os[NSTREAMS];
static hipStream_t stream[NSTREAMS];
static int ndevices;

static size_t d_indatasize; // size in bytes of non-uniform data
static size_t d_outdatasize; // size in bytes of gridded data
static size_t d_coilimgsize; // multi-coil image size
static size_t d_imgsize; // coil-combined image size
static size_t d_gridsize;
static size_t h_outdatasize;


#define DPRINT if(flag_verbose)printf
#define dprint(expr,fmt)  do{ if(flag_verbose)printf(#expr " = %" #fmt "\n", expr); }while(0);


// non-uniform data shape: nchan x nrep x nro x npe
// uniform data shape:     nchan x nrep x ngrid x ngrid x nz
// image shape:            nchan x nrep x nimg x nimg x nz
// coil-combined image:            nrep x nimg x nimg x nz



void
TRON_set_default_plan (TRON_plan *p)
{
    DPRINT("TRON_set_default_plan\n");

    // TODO: REMOVE THESE
    p->nchan = 0;
    p->nrep = 0;  // # of repeated measurements of same trajectory
    p->nro = 0;
    p->npe = 0;
    p->ngrid = 0;
    p->nx = 0;
    p->ny = 0;
    p->nz = 0;

    for (int i = 0; i < 5; ++i) {
        p->in_dims.n[i] = 0;
        p->out_dims.n[i] = 0;
    }

    // STYLE PARAMETERS
    p->prof_slide = 0;
    p->skip_angles = 0;
    p->prof_per_image = 0;
    p->grid_oversamp = 2.f;
    p->data_undersamp = 1.f;
    p->kernwidth = 2.f;
    p->nchunks = 1;;    // # of chunks to divide input data into for recon

    // BOOLEAN OPTIONS
    p->flags.adjoint = 0;
    p->flags.postcomp = 0;
    p->flags.deapodize = 1;
    p->flags.golden_angle = 0;
}

// CONSTANTS
static const float PHI = 1.9416089796736116f;

inline void
gpuAssert (hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code); }
    }
}
#define cuTry(ans) { gpuAssert((ans), __FILE__, __LINE__); }

static const char *
_cudaGetErrorEnum(hipfftResult error)
{
    switch (error) {
        case HIPFFT_SUCCESS: return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN: return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED: return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE: return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED: return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED: return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE: return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";
        default: return "<unknown>";
    }
}

#define cufftSafeCall(err)  __cufftSafeCall(err, __FILE__, __LINE__)
inline void __cufftSafeCall (hipfftResult err, const char *file, const int line)
{
    if (HIPFFT_SUCCESS != err) {
        fprintf(stderr, "CUFFT error in file '%s', line %d\nerror %s: %d\nterminating!\n",__FILE__, __LINE__, \
                _cudaGetErrorEnum(err), (int)err);
        hipDeviceReset();
        exit(1);
    }
}

__global__ void
fftshift (float2 *dst, const int n, const int nchan)
{
    float2 tmp;
    int dn = n / 2;
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < dn*dn; id += blockDim.x * gridDim.x)
    {
        int x = id / dn;
        int y = id % dn;
        int id1 = x*n + y;
        int id2 = (x + dn)*n + y;
        int id3 = (x + dn)*n + y + dn;
        int id4 = x*n + y + dn;
        for (int c = 0; c < nchan; ++c) {
            tmp = dst[id1*nchan + c]; // 1 <-> 3
            dst[id1*nchan + c] = dst[id3*nchan + c];
            dst[id3*nchan + c] = tmp;
            tmp = dst[id2*nchan + c]; // 2 <-> 4
            dst[id2*nchan + c] = dst[id4*nchan + c];
            dst[id4*nchan + c] = tmp;
        }
    }
}


__host__ void
fft_init(hipfftHandle *plan, const int nx, const int ny, const int nchan)
{
  DPRINT("fft_init\n");
  dprint(nx,d);
  dprint(ny,d);
  dprint(nchan,d);
  // setup FFT
  const int rank = 2;
  int idist = 1, odist = 1, istride = nchan, ostride = nchan;
  int n[2] = {nx, ny};
  int inembed[]  = {nx, ny};
  int onembed[]  = {nx, ny};
  cufftSafeCall(hipfftPlanMany(plan, rank, n, onembed, ostride, odist,
      inembed, istride, idist, HIPFFT_C2C, nchan));
}


__host__ void
fftwithshift (float2 *x, hipfftHandle plan, const int j, const int n, const int nrep)
{
    fftshift<<<gridsize,blocksize,0,stream[j]>>>(x, n, nrep);
    cufftSafeCall(hipfftExecC2C(plan, x, x, HIPFFT_FORWARD));
    fftshift<<<gridsize,blocksize,0,stream[j]>>>(x, n, nrep);
}

__host__ void
ifftwithshift (float2 *x, hipfftHandle plan, const int j, const int n, const int nrep)
{
    fftshift<<<gridsize,blocksize,0,stream[j]>>>(x, n, nrep);
    cufftSafeCall(hipfftExecC2C(plan, x, x, HIPFFT_BACKWARD));
    fftshift<<<gridsize,blocksize,0,stream[j]>>>(x, n, nrep);
}

__device__ void
powit (float2 *A, const int n, const int niters)
{
    /* replace first column of square matrix A with largest eigenvector */
    float2 x[MAXCHAN], y[MAXCHAN];
    for (int k = 0; k < n; ++k)
        x[k] = make_float2(1.f, 0.f);
    for (int t = 0; t < niters; ++t) {
        for (int j = 0; j < n; ++j) {
            y[j] = make_float2(0.f,0.f);
            for (int k = 0; k < n; ++k)
               y[j] += A[j*n + k]*x[k];
        }
        // calculate the length of the resultant vector
        float norm_sq = 0.f;
        for (int k = 0; k < n; ++k)
          norm_sq += norm(y[k]);
        norm_sq = sqrtf(norm_sq);
        for (int k = 0; k < n; ++k)
            x[k] = y[k] / norm_sq;
    }
    float2 lambda = make_float2(0.f,0.f);
    for (int j = 0; j < n; ++j) {
        y[j] = make_float2(0.f,0.f);
        for (int k = 0; k < n; ++k)
           y[j] += A[j*n + k]*x[k];
        lambda += conj(x[j])*y[j];
    }
    for (int j = 0; j < n; ++j)
        A[j] = x[j];
    A[n] = lambda;  // store dominant eigenvalue in A
}

__global__ void
coilcombinesos (float2 *img, const float2 * __restrict__ coilimg, const int nimg, const int nchan)
{
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < nimg*nimg; id += blockDim.x * gridDim.x) {
        float val = 0.f;
        for (int c = 0; c < nchan; ++c)
            val += norm(coilimg[nchan*id + c]);
        img[id].x = sqrtf(val);
        img[id].y = 0.f;
    }
}

__global__ void
coilcombinewalsh (float2 *img, const float2 * __restrict__ coilimg,
   const int nimg, const int nchan, const int npatch)
{
    float2 A[MAXCHAN*MAXCHAN];
    for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < nimg*nimg; id += blockDim.x * gridDim.x)
    {
        if (nchan == 1)
            img[id] = coilimg[id];
        else {
            int x = id / nimg;
            int y = id % nimg;
            for (int k = 0; k < NCHAN*NCHAN; ++k)
                A[k] = make_float2(0.f,0.f);
            for (int px = max(0,x-npatch); px <= min(nimg-1,x+npatch); ++px)
                for (int py = max(0,y-npatch); py <= min(nimg-1,y+npatch); ++py) {
                    int offset = nchan*(px*nimg + py);
                    for (int c2 = 0; c2 < nchan; ++c2)
                        for (int c1 = 0; c1 < nchan; ++c1)
                            A[c1*nchan + c2] += coilimg[offset+c1]*conj(coilimg[offset+c2]);
                }
            powit(A, nchan, 5);
            img[id] = make_float2(0.f, 0.f);
            for (int c = 0; c < nchan; ++c)
                img[id] += conj(A[c])*coilimg[nchan*id+c]; // * cexpf(-maxphase);
        }
// #ifdef CALC_B1
//         for (int c = 0; c < NCHAN; ++c) {
//             d_b1[nchan*id + c] = sqrtf(s[0])*U[nchan*c];
//         }
// #endif
    }
}

__host__ __device__ float
i0f (const float x)
{
    if (x == 0.f) return 1.f;
    float z = x * x;
    float num = (z* (z* (z* (z* (z* (z* (z* (z* (z* (z* (z* (z* (z*
        (z* 0.210580722890567e-22  + 0.380715242345326e-19 ) +
        0.479440257548300e-16) + 0.435125971262668e-13 ) +
        0.300931127112960e-10) + 0.160224679395361e-7  ) +
        0.654858370096785e-5)  + 0.202591084143397e-2  ) +
        0.463076284721000e0)   + 0.754337328948189e2   ) +
        0.830792541809429e4)   + 0.571661130563785e6   ) +
        0.216415572361227e8)   + 0.356644482244025e9   ) +
        0.144048298227235e10);
    float den = (z*(z*(z-0.307646912682801e4)+
        0.347626332405882e7)-0.144048298227235e10);
    return -num/den;
}

__host__ __device__ inline float
gridkernel (const float dx, const float dy, const float kernwidth, const float grid_oversamp)
{
    float r2 = dx*dx + dy*dy;
#ifdef KERN_KB
    //const float kernwidth = 2.f;
#define SQR(x) ((x)*(x))
#define BETA (M_PI*sqrtf(SQR(kernwidth/grid_oversamp*(grid_oversamp-0.5))-0.8))
    return r2 < kernwidth*kernwidth ? i0f(BETA * sqrtf (1.f - r2/kernwidth/kernwidth)) / i0f(BETA): 0.f;
#else
    const float sigma = 0.33f; // ballparked from Jackson et al. 1991. IEEE TMI, 10(3), 473–8
    return expf(-0.5f*r2/sigma/sigma);
#endif
}

// TODO: eliminate or combine the kernels
__host__ __device__ inline float
degridkernel (const float dx, const float dy, const float kernwidth, const float grid_oversamp)
{
    float r2 = dx*dx + dy*dy;
#ifdef KERN_KB
    //const float kernwidth = 2.f;
#define SQR(x) ((x)*(x))
#define BETA (M_PI*sqrtf(SQR(kernwidth/grid_oversamp*(grid_oversamp-0.5))-0.8))
    return r2 < kernwidth*kernwidth ? i0f(BETA * sqrtf (1.f - r2/kernwidth/kernwidth)) / i0f(BETA): 0.f;
#else
    const float sigma = 0.33f; // ballparked from Jackson et al. 1991. IEEE TMI, 10(3), 473–8
    return expf(-0.5f*r2/sigma/sigma);
#endif
}

__device__ inline float
modang (const float x)   /* rescale arbitrary angles to [0,2PI] interval */
{
    const float TWOPI = 2.f*M_PI;
    float y = fmodf(x, TWOPI);
    return y < 0.f ? y + TWOPI : y;
}

__device__ inline float
minangulardist(const float a, const float b)
{
    float d1 = fabsf(modang(a - b));
    float d2 = fabsf(modang(a + M_PI) - b);
    float d3 = 2.f*M_PI - d1;
    float d4 = 2.f*M_PI - d2;
    return fminf(fminf(d1,d2),fminf(d3,d4));
}

__host__ void
fillapod (float2 *d_apod, const int nx, const int ny, const float kernwidth, const float grid_oversamp)
{
    DPRINT("fillapod\n");
    const size_t d_imgsize = nx*ny*sizeof(float2);
    float2 *h_apod = (float2*)malloc(d_imgsize);
    int w = int(kernwidth);
    int n = ny;  // TODO: fix this, substitute correct dims

    for (int k = 0; k < n*n; ++k)
        h_apod[k] = make_float2(0.f,0.f);
    for (int x = 0; x < w; ++x) {
        for (int y = 0; y < w; ++y)
            h_apod[n*x + y].x = gridkernel(x, y, kernwidth, grid_oversamp);
        for (int y = n-w; y < n; ++y)
            h_apod[n*x + y].x = gridkernel(x, n-y, kernwidth, grid_oversamp);
    }
    for (int x = n-w; x < n; ++x) {
        for (int y = 0; y < w; ++y)
            h_apod[n*x + y].x = gridkernel(n-x, y, kernwidth, grid_oversamp);
        for (int y = n-w; y < n; ++y)
            h_apod[n*x + y].x = gridkernel(n-x, n-y, kernwidth, grid_oversamp);
    }
    cuTry(hipMemcpy(d_apod, h_apod, d_imgsize, hipMemcpyHostToDevice));
    hipfftHandle fft_plan_apod;
    cufftSafeCall(hipfftPlan2d(&fft_plan_apod, n, n, HIPFFT_C2C));
    cufftSafeCall(hipfftExecC2C(fft_plan_apod, d_apod, d_apod, HIPFFT_BACKWARD));
    fftshift<<<n,n>>>(d_apod, n, 1);
    cuTry(hipMemcpy(h_apod, d_apod, d_imgsize, hipMemcpyDeviceToHost));

    float maxval = 0.f;
    for (int k = 0; k < n*n; ++k) { // take magnitude and find brightest pixel at same time
        float mag = abs(h_apod[k]);
        h_apod[k] = make_float2(mag);
        maxval = mag > maxval ? mag : maxval;
    }
    for (int k = 0; k < n*n; ++k) { // normalize it   TODO: check for image artifacts
        h_apod[k].x /= maxval;
        h_apod[k].x = h_apod[k].x > 0.1f ? 1.0f / h_apod[k].x : 1.0f;
    }
    cuTry(hipMemcpy(d_apod, h_apod, d_imgsize, hipMemcpyHostToDevice));
    free(h_apod);
}

__global__ void
deapodize (float2 *img, const float2 * __restrict__ apod, const int nx, const int ny, const int nchan)
{
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < nx*ny; id += blockDim.x * gridDim.x)
        for (int c = 0; c < nchan; ++c)
            img[nchan*id+c] *= apod[id].x; // took magnitude prior
}

__global__ void  // TODO: fix this
degrid_deapodize (float2 *img, const int nimg, const int nchan,
    float kernwidth, float grid_oversamp)
{
    grid_oversamp = 1.f;
    kernwidth = 1.f;
    float beta = kernwidth*(grid_oversamp-0.5)/grid_oversamp;
    beta = M_PI*sqrtf(beta*beta - 0.8);
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < nimg*nimg; id += blockDim.x * gridDim.x)
    {
        int y = id % nimg - nimg/2;
        int x = id / nimg - nimg/2;
        float r = sqrtf(x*x + y*y);
        float d = M_PI*kernwidth*r/nimg;
        float s = d > beta ? sqrtf(d*d - beta*beta) : 1.f;
        float f = s != 0.f ? sinf(s) / s : 1.f;
        for (int c = 0; c < nchan; ++c)
            img[nchan*id+c] /= f;
    }
}


//__device__ float
//degrid_deapodize (const float r, const int ngrid, const float kernwidth, const float grid_oversamp)
//{
//#define SQR(x) ((x)*(x))
//#define BETA (M_PI*sqrtf(SQR(kernwidth/grid_oversamp*(grid_oversamp-0.5))-0.8))
    //float a = M_PI*kernwidth*r/float(ngrid);
    //float y = sqrtf(a*a - BETA*BETA);
    //float w = sinf(y) / y;
    //return w == 0.f ? 1.f : w;
//}


__global__ void
precompensate (float2 *nudata, const int nchan, const int nro, const int npe_per_image, const int nrest)
{
    float a = (2.f  - 2.f / float(npe_per_image)) / float(nro);
    float b = 1.f / float(npe_per_image);
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < nrest; id += blockDim.x * gridDim.x)
        for (int r = 0; r < nro; ++r) {
            float sdc = a*fabsf(r - float(nro/2)) + b;
            for (int c = 0; c < nchan; ++c)
                nudata[nro*nchan*id + nchan*r + c] *= sdc;
        }
}

__global__ void
crop (float2* dst, const int nxdst, const int nydst, const float2* __restrict__ src, const int nxsrc, const int nysrc, const int nchan)
{
    const int nsrc = nxsrc, ndst = nxdst;  // TODO: eliminate this
    const int w = (nsrc - ndst) / 2;
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < ndst*ndst; id += blockDim.x * gridDim.x)
    {
        int xdst = id / ndst;
        int ydst = id % ndst;
        int srcid = (xdst + w)*nsrc + ydst + w;
        for (int c = 0; c < nchan; ++c)
            dst[nchan*id + c] = src[nchan*srcid + c];
    }
}

// TODO: eliminate this
__global__ void
copy (float2* dst, const float2* __restrict__ src, const int n)
{
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < n; id += blockDim.x * gridDim.x)
        dst[id] = src[id];
}


__global__ void
pad (float2* dst, const int ndst, const float2* __restrict__ src, const int nsrc, const int nchan)
{
    // set whole array to zero first (not most efficient!)
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < ndst*ndst; id += blockDim.x * gridDim.x)
        for (int c = 0; c < nchan; ++c)
            dst[nchan*id + c] = make_float2(0.f, 0.f);
    // insert src into center of dst
    const int w = (ndst - nsrc) / 2;
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < nsrc*nsrc; id += blockDim.x * gridDim.x)
    {
        int xdst = id / nsrc;
        int ydst = id % nsrc;
        int dstid = (xdst + w)*nsrc + ydst + w;
        for (int c = 0; c < nchan; ++c)
            dst[nchan*dstid + c] = src[nchan*id + c];
    }
}

extern "C" {  // don't mangle name, so can call from other languages

/*
    grid a single 2D image from input radial data
*/
__global__ void
gridradial2d (float2 *udata, const float2 * __restrict__ nudata, const int ngrid,
    const int nchan, const int nro, const int npe, const float kernwidth, const float grid_oversamp,
const int skip_angles, const int flag_postcomp, const int flag_golden_angle)
{
    // udata: [NCHAN x NGRID x NGRID], nudata: NCHAN x NRO x NPE
    //float grid_oversamp = float(ngrid) / float(nro); // grid_oversampling factor
    float2 utmp[MAXCHAN];
    const int blocksizex = 8; // TODO: optimize this blocking
    const int blocksizey = 4;
    const int warpsize = blocksizex*blocksizey;
    //int nblockx = ngrid / blocksizex;
    int nblocky = ngrid / blocksizey; // # of blocks along y dimension
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < ngrid*ngrid; tid += blockDim.x * gridDim.x)
    {
        for (int ch = 0; ch < nchan; ch++)
          utmp[ch] = make_float2(0.f,0.f);

        //int x = id / ngrid - ngrid/2;
        //int y = -(id % ngrid) + ngrid/2;
        int z = tid / warpsize; // not a real z, just a block label
        int bx = z / nblocky;
        int by = z % nblocky;
        int zid = tid % warpsize;
        int x = zid / blocksizey + blocksizex*bx;
        int y = zid % blocksizey + blocksizey*by;
        int id = x*ngrid + y; // computed linear array index for uniform data
        x = -x + ngrid/2;
        y -= ngrid/2;
        float gridpoint_radius = hypotf(float(x), float(y));
        int rmax = fminf(floorf(gridpoint_radius + kernwidth)/grid_oversamp, nro/2-1);
        int rmin = fmaxf(ceilf(gridpoint_radius - kernwidth)/grid_oversamp, 0);  // define a circular band around the uniform point
        for (int ch = 0; ch < nchan; ++ch)
             udata[nchan*id + ch] = make_float2(0.f,0.f);
        if (rmin > nro/2-1) continue; // outside non-uniform data area

        float sdc = 0.f;
        // get uniform point coordinate in non-uniform system, (r,theta) in this case
        float gridpoint_theta = modang(atan2f(float(y),float(x)));
        float dtheta = atan2f(kernwidth, gridpoint_radius); // narrow that band to an arc
        // profiles must line within an arc of 2*dtheta to be counted

        // TODO: replace this logic with boolean function that can be swapped out
        // for diff acquisitions
        for (int pe = 0; pe < npe; ++pe)
        {
            float profile_theta = flag_golden_angle ? modang(PHI * float(pe + skip_angles)) : float(pe) * M_PI / float(npe) + M_PI/2;
            //float dtheta1 = fabsf(modang(profile_theta - gridpoint_theta));
            //float dtheta2 = fabsf(modang(profile_theta + M_PI) - gridpoint_theta);
            //float dtheta1 = fabsf(profile_theta - gridpoint_theta);
            //float dtheta2 = fabsf(profile_theta + M_PI - gridpoint_theta);
            //float dtheta3 = 2.f*M_PI - dtheta1;
            //float dtheta4 = 2.f*M_PI - dtheta2;
            float dtheta1 = minangulardist(profile_theta, gridpoint_theta);
            if (dtheta1 <= dtheta) // || dtheta2 <= dtheta || dtheta3 <= dtheta || dtheta4 <= dtheta)
            {
                float sf, cf;
                __sincosf(profile_theta, &sf, &cf);
                sf *= grid_oversamp;
                cf *= grid_oversamp;
                // TODO: fix this logic, try using without dtheta1
                //int rstart = dtheta1 <= dtheta || dtheta3 <= dtheta ? rmin : -rmax;
                //int rend   = dtheta1 <= dtheta || dtheta3 <= dtheta ? rmax : -rmin;
                int rstart = fabs(profile_theta-gridpoint_theta) < 0.5f*M_PI ? rmin : -rmax;
                int rend   = fabs(profile_theta-gridpoint_theta) < 0.5f*M_PI ? rmax : -rmin;
                for (int r = rstart; r <= rend; ++r)  // for each POSITIVE non-uniform ro point
                for (int r = rstart; r <= rend; ++r)  // for each POSITIVE non-uniform ro point
                {
                    float kx = r*cf; // [-NGRID/2 ... NGRID/2-1]    // TODO: compute distance in radial coordinates?
                    float ky = r*sf; // [-NGRID/2 ... NGRID/2-1]
                    float wgt = gridkernel(kx - x, ky - y, kernwidth, grid_oversamp);
                    if (flag_postcomp)
                      sdc += wgt;
                    for (int ch = 0; ch < nchan; ch++) { // unrolled by 2 'cuz faster
                        //utmp[ch] += wgt*nudata[nchan*(nro*pe + r + nro/2) + ch];
                        //utmp[ch + 1] += wgt*nudata[nchan*(nro*pe + r + nro/2) + ch + 1];
                        utmp[ch].x = __fmaf_rn(wgt,nudata[nchan*(nro*pe + r + nro/2) + ch].x, utmp[ch].x);
                        utmp[ch].y = __fmaf_rn(wgt,nudata[nchan*(nro*pe + r + nro/2) + ch].y, utmp[ch].y);
                    }
                }
            }
        }
        if (flag_postcomp && sdc > 0.f)
            for (int ch = 0; ch < nchan; ++ch)
                udata[nchan*id + ch] = utmp[ch] / sdc;
        else
            for (int ch = 0; ch < nchan; ++ch)
                udata[nchan*id + ch] = utmp[ch];
    }
}

/*  generate 2D radial data from an input 2D image */
__global__ void
degridradial2d (
    float2 *nudata, const float2 * __restrict__ udata, const int nimg,
    const int nchan, const int nro, const int npe, const float kernwidth,
    const float grid_oversamp, const int skip_angles, const int flag_golden_angle)
{
    // udata: [NCHAN x NGRID x NGRID], nudata: NCHAN x NRO x NPE
    //float grid_oversamp = float(ngrid) / float(nro); // grid_oversampling factor
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < nro*npe; id += blockDim.x * gridDim.x)
    {
        int pe = id / nro; // find my location in the non-uniform data
        int ro = id % nro;
        float r = (ro - 0.5f * nro )/ (float)(nro); // [-0.5,0.5-1/nro] convert indices to (r,theta) coordinates
        float t = flag_golden_angle ? modang(PHI*(pe + skip_angles)) : float(pe) * M_PI / float(npe)+ M_PI/2;
        float kx = r*cos(t); // [-0.5,0.5-1/nro] Cartesian freqs of non-Cart datum  // TODO: _sincosf?
        float ky = r*sin(t); // [-0.5,0.5-1/nro]
        float x = nimg*(0.5 - kx);  // [0,ngrid] (x,y) coordinates in grid units
        float y = nimg*(ky + 0.5);

        for (int ch = 0; ch < nchan; ++ch) // zero my elements
             nudata[nchan*id + ch] = make_float2(0.f, 0.f);
        for (int ux = fmaxf(0.f,x-kernwidth); ux <= fminf(nimg-1,x+kernwidth); ++ux)
        for (int uy = fmaxf(0.f,y-kernwidth); uy <= fminf(nimg-1,y+kernwidth); ++uy)
        {
            float wgt = degridkernel(ux - x, uy - y, kernwidth, grid_oversamp);
            for (int ch = 0; ch < nchan; ++ch) {
                float2 c = udata[nchan*(ux*nimg + uy) + ch] / (nro*npe*kernwidth*kernwidth); // TODO: check this
                nudata[nchan*id + ch].x += wgt*c.x;
                nudata[nchan*id + ch].y += wgt*c.y;
            }
        }
    }
}


void
tron_init (TRON_plan *p)
{

  uint64_t ngridx = p->out_dims.x*p->grid_oversamp;
  uint64_t ngridy = p->out_dims.y*p->grid_oversamp;

  if (MULTI_GPU) {
    cuTry(hipGetDeviceCount(&ndevices));
  } else
    ndevices = 1;
  DPRINT("MULTI_GPU = %d\n", MULTI_GPU);
  DPRINT("NSTREAMS = %d\n", NSTREAMS);
  DPRINT("Using %d CUDA devices\n", ndevices);
  DPRINT("Kernels configured with %d blocks of %d threads\n", gridsize, blocksize);

  // array sizes
  // TODO: this is wrong.  d_indatasize should just be the size of the work slice
  d_indatasize = dim_prod(&p->in_dims)*sizeof(float2) / p->nchunks;  // input data
  d_outdatasize = dim_prod(&p->out_dims)*sizeof(float2) / p->nchunks; // multi-coil gridded data

  // TODO: eliminate these by simplifying data storage
  d_gridsize = ngridx*ngridy*sizeof(float2);  // single channel grid size
  d_coilimgsize = p->in_dims.c*p->out_dims.x*p->out_dims.y*sizeof(float2); // coil images
  d_imgsize = p->out_dims.x*p->out_dims.y*sizeof(float2); // coil-combined image

  dprint(d_indatasize,ld);
  dprint(d_outdatasize,ld);
  dprint(d_gridsize,ld);
  dprint(d_coilimgsize,ld);
  dprint(d_imgsize,ld);

  for (int j = 0; j < NSTREAMS; ++j) // allocate data and initialize apodization and kernel texture
  {
      DPRINT("init STREAM %d\n", j);
      if (MULTI_GPU) hipSetDevice(j % ndevices);
      cuTry(hipStreamCreate(&stream[j]));

      fft_init(&fft_plan[j], p->out_dims.x, p->out_dims.y, p->out_dims.c);
      cufftSafeCall(hipfftSetStream(fft_plan[j], stream[j]));

      fft_init(&fft_plan_os[j], ngridx, ngridy, p->out_dims.c);
      cufftSafeCall(hipfftSetStream(fft_plan_os[j], stream[j]));

      cuTry(hipMalloc((void **)&d_indata[j],  d_indatasize));
      cuTry(hipMalloc((void **)&d_outdata[j], d_outdatasize));
      cuTry(hipMalloc((void **)&d_tmp[j],     d_outdatasize));

      // // cuTry(hipMemset(d_outdata[j], 0, p->d_outdatasize));
      // cuTry(hipMalloc((void **)&d_coilimg[j], d_coilimgsize));
      // //cuTry(hipMalloc((void **)&d_b1[j], d_coilimgsize));
      // cuTry(hipMalloc((void **)&d_img[j], d_imgsize));

      // TODO: only fill apod if depapodize is called
      // TODO: handle adjoint vs non-adjoint
      cuTry(hipMalloc((void **)&d_apodos[j], d_gridsize));
      cuTry(hipMalloc((void **)&d_apod[j], d_imgsize));
      fillapod(d_apodos[j], p->out_dims.x, p->out_dims.y, p->kernwidth, p->grid_oversamp);
      crop<<<p->nimg,p->nimg>>>(d_apod[j], p->out_dims.x, p->out_dims.y, d_apodos[j], ngridx, ngridy, 1);
      cuTry(hipFree(d_apodos[j]));

  }
}

void
tron_shutdown()
{
    DPRINT("freeing device memory\n");
    for (int j = 0; j < NSTREAMS; ++j) { // free allocated memory
        if (MULTI_GPU) hipSetDevice(j % ndevices);
        cuTry(hipFree(d_indata[j]));
        cuTry(hipFree(d_outdata[j]));
        cuTry(hipFree(d_tmp[j]));

        // cuTry(hipFree(d_coilimg[j]));
        // //cuTry(hipFree(d_b1[j]));
        // cuTry(hipFree(d_img[j]));
        cuTry(hipFree(d_apod[j]));
        hipStreamDestroy(stream[j]);
    }
}


/*  Reconstruct images from 2D radial data.  This host routine calls the appropriate
    CUDA kernels in the correct order depending on the direction of recon.   */

__host__ void
recon_radial_2d(float2 *h_outdata, const float2 *__restrict__ h_indata, TRON_plan *p)
{
    DPRINT("recon_radial_2d\n");
    tron_init(p);
    int ngridx = p->out_dims.x * p->grid_oversamp;
    int ngridy = p->out_dims.y * p->grid_oversamp;
    dprint(p->out_dims.z, d);

    for (int z = 0; z < p->out_dims.z; ++z)
    {
        int j = z % NSTREAMS; // j is stream index
        if (MULTI_GPU) hipSetDevice(j % ndevices);

        int peoffset = z*p->prof_slide;

        // address offsets into the data arrays
        size_t data_offset = p->in_dims.c * p->in_dims.r * peoffset;
        size_t img_offset = p->out_dims.x * p->out_dims.y * z;

        printf("[dev %d, stream %d] reconstructing slice %d/%d from PEs %d-%d (offset %ld)\n",
            j%ndevices, j, z+1, p->out_dims.z, z*p->prof_slide, (z+1)*p->prof_slide-1, data_offset);

        dprint(img_offset,ld);
        dprint(data_offset,ld);
        dprint(d_indatasize,ld);
        dprint(stream[j], ld);
        cuTry(hipMemcpyAsync(d_indata[j], h_indata + data_offset, d_indatasize, hipMemcpyHostToDevice, stream[j]));
        DPRINT("input data copied\n");

        if (p->flags.adjoint)
        {
            DPRINT("performing ADJOINT\n");
            // reverse from non-uniform data to image
            precompensate<<<gridsize,blocksize,0,stream[j]>>>(d_indata[j], p->in_dims.c, p->in_dims.r, p->prof_per_image, p->in_dims.theta);
            gridradial2d<<<gridsize,blocksize,0,stream[j]>>>(d_outdata[j], d_indata[j],
                ngridx, p->in_dims.c, p->in_dims.r, p->prof_per_image, p->kernwidth,
                p->grid_oversamp, p->skip_angles+peoffset, p->flags.postcomp, p->flags.golden_angle);
            ifftwithshift(d_outdata[j], fft_plan_os[j], j, ngridx, p->out_dims.t*p->out_dims.c);
            crop<<<gridsize,blocksize,0,stream[j]>>>(d_tmp[j], p->out_dims.x, p->out_dims.y, d_outdata[j], ngridx, ngridy, p->in_dims.c);
            // TODO: look at in_dims.c vs out_dims.c to decide whether to coil combine and by how much (can compress)
            coilcombinewalsh<<<gridsize,blocksize,0,stream[j]>>>(d_outdata[j], d_tmp[j], p->out_dims.x, p->in_dims.c, 1); /* 0 works, 1 good, 3 better */
            //coilcombinesos<<<gridsize,blocksize,0,stream[j]>>>(d_outdata[j], d_tmp[j], nimg, nchan);
            deapodize<<<gridsize,blocksize,0,stream[j]>>>(d_outdata[j], d_apod[j], p->out_dims.x, p->out_dims.y, p->out_dims.c);
            dprint(img_offset,ld);
            cuTry(hipMemcpyAsync(h_outdata + img_offset, d_outdata[j], d_imgsize, hipMemcpyDeviceToHost, stream[j]));
        }
        else
        {   // forward from image to non-uniform data
            DPRINT("performing FORWARD\n");
            degrid_deapodize<<<gridsize,blocksize,0,stream[j]>>>(d_indata[j], p->in_dims.x, 1, p->kernwidth, p->grid_oversamp);
            fftwithshift(d_indata[j], fft_plan_os[j], j, p->in_dims.x, p->in_dims.t*p->in_dims.c);
            //copy<<<gridsize,blocksize,0,stream[j]>>>(d_indata[j], d_img[j], nimg*nimg);
            degridradial2d<<<gridsize,blocksize,0,stream[j]>>>(d_outdata[j], d_indata[j],
                p->in_dims.x, p->in_dims.c, p->out_dims.r, p->out_dims.theta, p->kernwidth, p->grid_oversamp, p->skip_angles, p->flags.golden_angle);
            cuTry(hipMemcpyAsync(h_outdata + p->out_dims.c*p->out_dims.r*p->out_dims.theta*z, d_indata[j], d_indatasize, hipMemcpyDeviceToHost, stream[j]));
        }

    }

    tron_shutdown();
}




}

void
print_usage()
{
    fprintf(stderr, "Usage: tron [-3ahuv] [-r cmds] [-d prof_slide] [-k width] [-o grid_oversamp] [-s skip_angles] [-u data_undersamp] <infile.ra> [outfile.ra]\n");
    fprintf(stderr, "\t-3\t\t\3D koosh ball trajectory\n");
    fprintf(stderr, "\t-a\t\t\tadjoint operation\n");
    fprintf(stderr, "\t-d prof_slide\t\t\tnumber of phase encodes to slide between slices for helical scans\n");
    fprintf(stderr, "\t-g\t\t\tgolden angle radial\n");
    fprintf(stderr, "\t-h\t\t\tshow this help\n");
    fprintf(stderr, "\t-k width\t\twidth of gridding kernel\n");
    fprintf(stderr, "\t-o grid_oversamp\t\tgrid grid oversampling factor\n");
    fprintf(stderr, "\t-r nro\t\t\tnumber of readout points\n");
    fprintf(stderr, "\t-s skip_angles\t\tnumber of initial phase encodes to skip\n");
    fprintf(stderr, "\t-u data_undersamp\t\tinput data undersampling factor\n");
    fprintf(stderr, "\t-v\t\t\tverbose output\n");
}


int
main (int argc, char *argv[])
{
    float2 *h_indata, *h_outdata;
    ra_t ra_in, ra_out;
    int c, index;
    char infile[1024], outfile[1024];

    TRON_plan p;
    TRON_set_default_plan(&p);

    opterr = 0;
    while ((c = getopt (argc, argv, "3ad:ghk:o:r:s:u:v")) != -1)
    {
        switch (c) {
            case '3':
                p.flags.koosh = 1;
            case 'a':
                p.flags.adjoint = 1;
                break;
            case 'd':
                p.prof_slide = atoi(optarg);
                break;
            case 'g':
                p.flags.golden_angle = 1;
                break;
            case 'h':
                print_usage();
                return 1;
            case 'k':
                p.kernwidth = atof(optarg);
                break;
            case 'o':
                p.grid_oversamp = atof(optarg);
                break;
            case 'u':
                p.data_undersamp = atof(optarg);
                break;
            case 'r':
                p.nro = atoi(optarg);
                break;
            case 's':
                p.skip_angles = atoi(optarg);
                break;
            case 'v':
                flag_verbose = 1;
                break;
            default:
                print_usage();
                return 1;
        }
    }

    // set input and output files
    snprintf(outfile, 1024, "img_tron.ra"); // default value
    if (argc == optind) {
       print_usage();
       return 1;
    }
    for (index = optind; index < argc; index++) {
      if (index == optind)
        snprintf(infile, 1024, "%s", argv[index]);
      else if (index == optind + 1)
        snprintf(outfile, 1024, "%s", argv[index]);
    }

    dprint(p.skip_angles, d);
    dprint(p.prof_slide, d);
    dprint(p.kernwidth, .1f);
    dprint(p.data_undersamp, .3f);
    dprint(p.grid_oversamp, .3f);
    dprint(infile, s)
    dprint(outfile, s)

    DPRINT("Reading %s\n", infile);
    ra_read(&ra_in, infile);
    h_indata = (float2*)ra_in.data;
    assert(ra_in.ndims == 5);
    memcpy(p.in_dims.n, ra_in.dims, 5*sizeof(uint64_t));
    DPRINT("Sanity check: indata[0] = %f + %f i\n", h_indata[0].x, h_indata[0].y);
    DPRINT("in_dims = {%lu, %lu, %lu, %lu, %lu}\n", p.in_dims.c, p.in_dims.t, p.in_dims.x, p.in_dims.y, p.in_dims.z);
    assert(p.in_dims.c % 2 == 0 || p.in_dims.c == 1); // only single or even dimensions implemented for now


    printf("WARNING: Assuming square Cartesian dimensions for now.\n");

    // HERE IS WHERE WE COMPUTE OUTPUT DIMENSIONS BASED ON INPUT AND OPTIONAL ARGS
    if (p.flags.adjoint)
    {
        p.out_dims.c = 1;
        p.out_dims.t = p.in_dims.t;
        p.out_dims.x = p.in_dims.r / 2;
        p.out_dims.y = p.in_dims.r / 2;
        p.prof_per_image = p.data_undersamp * p.in_dims.r;  // TODO: fix this hack
        if (p.flags.koosh)
            p.out_dims.z = p.in_dims.r / 2;
        else
            p.out_dims.z = (p.in_dims.y - p.prof_per_image) / p.prof_slide;
    }
    else
    {
        p.prof_per_image = p.data_undersamp * p.in_dims.x;  // TODO: fix this hack
        p.out_dims.c = 1;
        p.out_dims.t = p.in_dims.t;
        p.out_dims.r = p.in_dims.x * 2;  // TODO: implement non-square images
        p.out_dims.theta = p.out_dims.r;   // TODO: make this more customizable
        p.out_dims.phi = p.flags.koosh ? p.out_dims.z : 1;
        p.prof_per_image = p.out_dims.theta;
    }
    p.nchunks = p.in_dims.n[3]*p.in_dims.n[4] / p.prof_per_image;
    dprint(p.nchunks,d);
    h_outdatasize = sizeof(float2);
    for (int k = 0; k < 5; ++k)
        h_outdatasize *= p.out_dims.n[k];


    // allocate pinned memory, which allows async calls
#ifdef CUDA_HOST_MALLOC
    cuTry(hipHostMalloc((void**)&h_outdata, h_outdatasize));
#else
    h_outdata = (float2*)malloc(h_outdatasize);
#endif


    DPRINT("Running reconstruction ...\n ");
    clock_t start = clock();

    recon_radial_2d(h_outdata, h_indata, &p);

    clock_t end = clock();
    DPRINT("Elapsed time: %.2f s\n", ((float)(end - start)) / CLOCKS_PER_SEC);

    DPRINT("Saving result to %s\n", outfile);
    ra_out.flags = 0;
    ra_out.eltype = 4;
    ra_out.elbyte = 8;
    ra_out.size = h_outdatasize;
    ra_out.ndims = 5;
    memcpy(ra_out.dims, p.out_dims.n, 5*sizeof(uint64_t));
    ra_out.data = (uint8_t*)h_outdata;
    ra_write(&ra_out, outfile);

    DPRINT("Cleaning up.\n");
    ra_free(&ra_in);
#ifdef CUDA_HOST_MALLOC
    //hipHostFree(&h_indata);
    hipHostFree(&h_outdata);
#else
    //free(h_indata);
    free(h_outdata);
#endif
    hipDeviceReset();

    return 0;
}
