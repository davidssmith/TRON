#include "hip/hip_runtime.h"
/*
  This file is part of the TRON package (http://github.com/davidssmith/TRON).

  The MIT License (MIT)

  Copyright (c) 2016-2017 David Smith

  Permission is hereby granted, free of charge, to any person obtaining a # copy
  of this software and associated documentation files (the "Software"), to # deal
  in the Software without restriction, including without limitation the # rights
  to use, copy, modify, merge, publish, distribute, sublicense, and/or # sell
  copies of the Software, and to permit persons to whom the Software is
  furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included # in all
  copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS # OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL # THE
  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING # FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS # IN THE
  SOFTWARE.
*/

#include <stdio.h>
#include <fcntl.h>
#include <unistd.h>
#include <err.h>
#include <errno.h>
#include <string.h>
#include <math.h>
#include <complex.h>
#include <time.h>
#include <stdint.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "float2math.h"
#include "mri.h"
#include "ra.h"
#include "tron.h"
#include "norm.h"

#define MAX(a,b) ((a)>(b)?(a):(b))
#define MIN(a,b) ((a)<(b)?(a):(b))
#define DPRINT if(flags.verbose)printf
#define dprint(expr,fmt)  do{ if(flags.verbose)fprintf(stderr,"\e[90m%d: " #expr " = %" #fmt "\e[0m\n", __LINE__, expr); }while(0);

// MISC GLOBAL VARIABLES
static hipfftHandle fft_plan[NSTREAMS], fft_plan_os[NSTREAMS];
static hipStream_t stream[NSTREAMS];
static int ndevices;

// DEVICE ARRAYS AND SIZES
static float2 *d_u[NSTREAMS], *d_v[NSTREAMS];
static size_t d_datasize; // size in bytes of non-uniform data
static size_t h_outdatasize;

// DEFAULT RECON CONFIGURATION
static float gridos = 2.f;  // TODO: compute ngrid from nx, ny and oversamp
static float kernwidth = 2.f;
static float data_undersamp = 1.f;

static int prof_slide = 0;         // # of profiles to slide through the data between reconstructed images
static int skip_angles = 0;        // # of angles to skip at beginning of image stack
static int peoffset = 0;

static int nc;  //  # of receive channels;
static int nt;  // # of repeated measurements of same trajectory
static int nro, npe1, npe2, npe1work;//, npe2work;  // radial readout and phase encodes
static int nx, ny, nz, nxos, nyos, nzos;  // Cartesian dimensions of uniform data

static struct {
    unsigned adjoint       : 1;
    unsigned deapodize     : 1;
    unsigned koosh         : 1;
    unsigned verbose       : 1;
    unsigned golden_angle  : 4;   // padded to 8 bits
} flags = {0, 1, 0, 0, 0};

// CONSTANTS
static const float PHI = 1.9416089796736116f;

inline void
gpuAssert (hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s in %s at L%d\n", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code); }
    }
}
#define cuTry(ans) { gpuAssert((ans), __FILE__, __LINE__); }

static const char *
_cufftGetErrorEnum(hipfftResult error)
{
    switch (error) {
        case HIPFFT_SUCCESS: return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN: return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED: return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE: return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED: return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED: return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE: return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";
        default: return "<unknown>";
    }
}

static const char *
_cublasGetErrorEnum (hipblasStatus_t error)
{
    switch (error) {
        case HIPBLAS_STATUS_SUCCESS: return "Success";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "Not Initiaziled";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "Alloc Failed";
        case HIPBLAS_STATUS_INVALID_VALUE: return "Invalid Value";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "Arch Mismatch";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "Mapping Error";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "Exec Failed";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "Internal Error";
        case HIPBLAS_STATUS_NOT_SUPPORTED: return "Not Supported";
        case HIPBLAS_STATUS_UNKNOWN: return "License Error";
        default: return "<unknown>";
    }
}

#define cufftSafeCall(err)  __cufftSafeCall(err, __FILE__, __LINE__)
inline void __cufftSafeCall (hipfftResult err, const char *file, const int line)
{
    if (HIPFFT_SUCCESS != err) {
        fprintf(stderr, "CUFFT error in file '%s', line %d\nerror %s: %d\nterminating!\n",__FILE__, __LINE__, \
                _cufftGetErrorEnum(err), (int)err);
        hipDeviceReset();
        exit(1);
    }
}

#define cublasSafeCall(err)  __cublasSafeCall(err, __FILE__, __LINE__)
inline void __cublasSafeCall (hipblasStatus_t err, const char *file, const int line)
{
    if (HIPBLAS_STATUS_SUCCESS != err) {
        fprintf(stderr, "CUBLAS error in file '%s', line %d\nerror %s: %d\nterminating!\n",__FILE__, __LINE__, \
                _cublasGetErrorEnum(err), (int)err);
        exit(1);
    }
}

enum fftshift_direction { FFT_SHIFT_FORWARD, FFT_SHIFT_INVERSE };

__global__ void
fftshift (float2 *dst, float2 *src, const int n, const int nchan, int direction)
{
    int offset = direction == FFT_SHIFT_FORWARD ? n/2 : n - n/2;

    for (int idsrc = blockIdx.x * blockDim.x + threadIdx.x; idsrc < n*n; idsrc += blockDim.x * gridDim.x)
    {
        int xsrc = idsrc / n;
        int ysrc = idsrc % n;
        int xdst = (xsrc + offset) % n;
        int ydst = (ysrc + offset) % n;
        int iddst = n*xdst + ydst;
        for (int c = 0; c < nchan; ++c) {
          dst[iddst*nchan + c].x = src[idsrc*nchan + c].x;
          dst[iddst*nchan + c].y = src[idsrc*nchan + c].y;
        }
    }
}


__global__ void
fftshift_inplace (float2 *dst, const int n, const int nchan)
{
    float2 tmp;
    int dn = n / 2;
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < dn*dn; id += blockDim.x * gridDim.x)
    {
        int x = id / dn;
        int y = id % dn;
        int id1 = x*n + y;
        int id2 = (x + dn)*n + y;
        int id3 = (x + dn)*n + y + dn;
        int id4 = x*n + y + dn;
        for (int c = 0; c < nchan; ++c) {
            tmp = dst[id1*nchan + c]; // 1 <-> 3
            dst[id1*nchan + c] = dst[id3*nchan + c];
            dst[id3*nchan + c] = tmp;
            tmp = dst[id2*nchan + c]; // 2 <-> 4
            dst[id2*nchan + c] = dst[id4*nchan + c];
            dst[id4*nchan + c] = tmp;
        }
    }
}

__host__ void
fft_init(hipfftHandle *plan, const int nx, const int ny, const int nchan)
{
  // setup FFT
  if (nchan == 1)
      cufftSafeCall(hipfftPlan2d(plan, nx, ny, HIPFFT_C2C));
  else {
      const int rank = 2;
      int idist = 1, odist = 1, istride = nchan, ostride = nchan;
      int n[2] = {nx, ny};
      int inembed[]  = {nx, ny};
      int onembed[]  = {nx, ny};
      cufftSafeCall(hipfftPlanMany(plan, rank, n, onembed, ostride, odist,
          inembed, istride, idist, HIPFFT_C2C, nchan));
  }
}

__device__ void
powit (float2 *A, const int n, const int niters)
{
    /* replace first column of square matrix A with largest eigenvector */
    float2 x[MAXCHAN], y[MAXCHAN];
    for (int k = 0; k < n; ++k)
        x[k] = make_float2(1.f, 0.f);
    for (int t = 0; t < niters; ++t) {
        for (int j = 0; j < n; ++j) {
            y[j] = make_float2(0.f,0.f);
            for (int k = 0; k < n; ++k)
               y[j] += A[j*n + k]*x[k];
        }
        // calculate the length of the resultant vector
        float norm_sq = 0.f;
        for (int k = 0; k < n; ++k)
          norm_sq += norm(y[k]);
        norm_sq = sqrtf(norm_sq);
        for (int k = 0; k < n; ++k)
            x[k] = y[k] / norm_sq;
    }
    float2 lambda = make_float2(0.f,0.f);
    for (int j = 0; j < n; ++j) {
        y[j] = make_float2(0.f,0.f);
        for (int k = 0; k < n; ++k)
           y[j] += A[j*n + k]*x[k];
        lambda += conj(x[j])*y[j];
    }
    for (int j = 0; j < n; ++j)
        A[j] = x[j];
    A[n] = lambda;  // store dominant eigenvalue in A
}

__global__ void
coilcombinesos (float2 *img, const float2 * __restrict__ coilimg, const int nimg, const int nchan)
{
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < nimg*nimg; id += blockDim.x * gridDim.x) {
        if (nchan > 1) {
          float val = 0.f;
          for (int c = 0; c < nchan; ++c)
              val += norm(coilimg[nchan*id + c]);
          img[id].x = sqrtf(val);
          img[id].y = 0.f;
        } else
          img[id] = coilimg[id];
    }
}

__global__ void
coilcombinewalsh (float2 *img, const float2 * __restrict__ coilimg,
   const int nimg, const int nchan, const int nt, const int npatch)
{
    float2 A[MAXCHAN*MAXCHAN];
    for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < nimg*nimg; id += blockDim.x * gridDim.x)
    {
        if (nchan == 1)
            img[id] = coilimg[id];
        else {
            int x = id / nimg;
            int y = id % nimg;
            for (int k = 0; k < NCHAN*NCHAN; ++k)
                A[k] = make_float2(0.f,0.f);
            for (int px = max(0,x-npatch); px <= min(nimg-1,x+npatch); ++px)
                for (int py = max(0,y-npatch); py <= min(nimg-1,y+npatch); ++py) {
                    int offset = nchan*(px*nimg + py);
                    for (int c2 = 0; c2 < nchan; ++c2)
                        for (int c1 = 0; c1 < nchan; ++c1)
                            A[c1*nchan + c2] += coilimg[offset+c1]*conj(coilimg[offset+c2]);
                }
            powit(A, nchan, 5);
            img[id] = make_float2(0.f, 0.f);
            for (int c = 0; c < nchan; ++c)
                img[id] += conj(A[c])*coilimg[nchan*id+c]; // * cexpf(-maxphase);
        }
// #ifdef CALC_B1
//         for (int c = 0; c < NCHAN; ++c) {
//             d_b1[nchan*id + c] = sqrtf(s[0])*U[nchan*c];
//         }
// #endif
    }
}

#if 1
__device__ float
besseli0 (const float x)
{
    if (x == 0.f) return 1.f;
    float z = x * x;
    float num = (z* (z* (z* (z* (z* (z* (z* (z* (z* (z* (z* (z* (z*
        (z* 0.210580722890567e-22  + 0.380715242345326e-19 ) +
        0.479440257548300e-16) + 0.435125971262668e-13 ) +
        0.300931127112960e-10) + 0.160224679395361e-7  ) +
        0.654858370096785e-5)  + 0.202591084143397e-2  ) +
        0.463076284721000e0)   + 0.754337328948189e2   ) +
        0.830792541809429e4)   + 0.571661130563785e6   ) +
        0.216415572361227e8)   + 0.356644482244025e9   ) +
        0.144048298227235e10);
    float den = (z*(z*(z-0.307646912682801e4)+
        0.347626332405882e7)-0.144048298227235e10);
    return -num/den;
}

#else

__host__ __device__ static float
besseli0 (float x)
{
    if (x < 0)  // TODO: is this correct?
       x = -x;
    float ax = fabsf(x);
    float ans;
    float y;

    if (ax < 3.75)
    {
        y=x/3.75,y=y*y;
        ans=1.0+y*(3.5156229+y*(3.0899424+y*(1.2067492
               +y*(0.2659732+y*(0.360768e-1+y*0.45813e-2)))));
    }
    else
    {
        y=3.75/ax;
        ans=(expf(ax)/sqrtf(ax))*(0.39894228+y*(0.1328592e-1
                +y*(0.225319e-2+y*(-0.157565e-2+y*(0.916281e-2
                +y*(-0.2057706e-1+y*(0.2635537e-1+y*(-0.1647633e-1
                +y*0.392377e-2))))))));
    }
    return ans;
}

#endif




__device__ inline float
kernel_shape (const float kernwidth, const float gridos)
{
//#define BEATTY_BETA
#ifdef BEATTY_BETA
    float a = kernwidth / gridos;
    float b = gridos - 0.5f;
    return M_PI*sqrtf(a*a*b*b - 0.8f);
#else
    //return M_PI*(2.f - 1.f/gridos);
    return 2.34f*2.0f*kernwidth;
#endif
}


__device__ inline float
gridkernel (const float x, const float kernwidth, const float sigma)
{
  // x in [-kernwidth,kernwidth]
  // from FesslerA
  //const float J = 2.0f*kernwidth;  // TODO: substitute kernwidth for J
  //const float alpha = 2.34f*J;
  float alpha = kernel_shape(kernwidth, sigma);
  if (fabsf(x) < kernwidth) {
      float r = x/kernwidth;
      float f = sqrtf(1.0f - r*r);
      return besseli0(alpha*f) / besseli0(alpha);
  } else
      return 0.0f;
}

__device__ inline float
gridkernelhat (const float u, const float kernwidth, const float sigma)
{
    // u in [-n/2,n/2]
    // from Fessler
    const float J = 2.0f*kernwidth;
    //const float alpha = 2.34f*J;
    float alpha = kernel_shape(kernwidth, sigma);
    //const int d = 1;
    float r = M_PI*J*u;
    float q = r*r - alpha*alpha;  // TODO: fix DomainError
    float y, z;
    if (q > 0) {
        z = sqrtf(q);
        y = J * sinf(z) / z / besseli0(alpha);
    } else if (q < 0) {
        z = sqrtf(-q);
        y = J * sinhf(z) / z / besseli0(alpha);
    } else
        y = 0.0f;
    // identity: J_1/2(z) = sin(z) * sqrt(2/pi/z)
    return y;
}

__device__ inline float
modang (const float x)   /* rescale arbitrary angles to [0,2PI] interval */
{
    const float TWOPI = 2.f*M_PI;
    float y = fmodf(x, TWOPI);
    return y < 0.f ? y + TWOPI : y;
}

__device__ inline float
minangulardist(const float a, const float b)
{
    float d1 = fabsf(modang(a - b));
    float d2 = fabsf(modang(a + M_PI) - b);
    float d3 = 2.f*M_PI - d1;
    float d4 = 2.f*M_PI - d2;
    return fminf(fminf(d1,d2),fminf(d3,d4));
}

__global__ void
deapodkernel (float2  *d_a, const int n, const int nrep, const float m, const float sigma)
{
    for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < n*n; id += blockDim.x * gridDim.x)
    {
        float x = id / float(n) - 0.5f*n;  // TODO: simplify this
        float y = float(id % n) - 0.5f*n;
        float scale = 1.f / (n * sigma);
        float r = sqrtf(x*x + y*y);
        //float wgt = gridkernelhat(x*scale, m, sigma) * gridkernelhat(y*scale, m, sigma);
        float wgt = gridkernelhat(r*scale, m, sigma);
        for (int c = 0; c < nrep; ++c)
            d_a[nrep*id + c] /= (wgt > 0.f ? wgt : 1.f);
    }
}


__global__ void
precompensate (float2 *nudata, const int nchan, const int nro, const int npe1work)
{
    float a = (2.f  - 2.f / float(npe1work)) / float(nro);
    float b = 1.f / float(npe1work);
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < npe1work; id += blockDim.x * gridDim.x)
        for (int r = 0; r < nro; ++r) {
            float sdc = a*fabsf(r - float(nro/2)) + b;
            for (int c = 0; c < nchan; ++c)
                nudata[nro*nchan*id + nchan*r + c] *= sdc;
        }
}

__global__ void
crop (float2* dst, const int nxdst, const int nydst, const float2* __restrict__ src, const int nxsrc, const int nysrc, const int nchan)
{
    const int nsrc = nxsrc, ndst = nxdst;  // TODO: eliminate this
    const int w = (nsrc - ndst) / 2;
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < ndst*ndst; id += blockDim.x * gridDim.x)
    {
        int xdst = id / ndst;
        int ydst = id % ndst;
        int srcid = (xdst + w)*nsrc + ydst + w;
        for (int c = 0; c < nchan; ++c)
            dst[nchan*id + c] = src[nchan*srcid + c];
    }
}


extern "C" {  // don't mangle name, so can call from other languages

/*
    grid a single 2D image from input radial data
*/
__global__ void
gridradial2d (float2 *udata, const float2 * __restrict__ nudata, const int nxos,
    const int nchan, const int nro, const int npe, const float kernwidth, const float gridos,
const int skip_angles, const int flag_golden_angle)
{
    // udata: [NCHAN x NGRID x NGRID], nudata: NCHAN x NRO x NPE
    float2 utmp[MAXCHAN];
    //const int blocksx = 8; // TODO: optimize this blocking
    //const int blocksy = 4;
    //const int warpsize = blocksx*blocksy;
    //int nblockx = nxos / blocksx;
    //int nblocky = nxos / blocksy; // # of blocks along y dimension

    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < nxos*nxos; id += blockDim.x * gridDim.x)
    {
        for (int ch = 0; ch < nchan; ch++)
            utmp[ch] = make_float2(0.f,0.f);

        // figure out this thread's Cartesian and radial coordinates
        int Y = id / nxos - nxos/2;
        int X = (id % nxos) - nxos/2;
        float R = hypotf(float(X), float(Y));

        // more complicated, but faster ... can probably optimize better by sorting, though
        // int z = tid / warpsize; // not a real z, just a block label
        // int by = z / nblocky;
        // int bx = z % nblocky;
        // int zid = tid % warpsize;
        // int y = zid / blocksy + blocksx*by;
        // int x = zid % blocksy + blocksy*bx;
        // int id = y*nxos + x; // computed linear array index for uniform data
        // x -= nxos/2;
        // y -= nxos/2;

        // define a circular band around the uniform point
        int Rmax = fminf(floorf(R + kernwidth), nxos/2-1);
        int Rmin = fmaxf(ceilf(R - kernwidth), 0);

        // zero the temporary work array
        // for (int ch = 0; ch < nchan; ++ch)
        //      udata[nchan*id + ch] = make_float2(0.f, 0.f);
        //if (Rmin > nxos/2-1) continue; // skip gridding if outside non-uniform data area

        // get uniform point coordinate in non-uniform system, (r,theta) in this case
        float T = modang(atan2f(float(Y),float(X)));
        float dT = atan2f(kernwidth, R); // narrow that band to an arc
        // profiles must lie within an arc of 2*dT to be counted

        // TODO: replace this logic with boolean function that can be swapped out
        // for diff acquisitions
        for (int pe = 0; pe < npe; ++pe)
        {
            float t = flag_golden_angle ? modang(PHI * float(pe + skip_angles)) : pe*M_PI / float(npe);
            float dt1 = minangulardist(t, T);
            if (dt1 <= dT)
            {
                float sf, cf;
                __sincosf(t, &sf, &cf);
                // TODO: fix this logic, try using without dt1
                int rstart = fabs(t-T) < 0.5f*M_PI ? Rmin : -Rmax;
                int rend   = fabs(t-T) < 0.5f*M_PI ? Rmax : -Rmin;
                for (int r = rstart; r <= rend; ++r)  // for each POSITIVE non-uniform ro point
                {
                    float kx = r*cf; // [-nxos/2 ... nxos/2-1]    // TODO: compute distance in radial coordinates?
                    float ky = r*sf; // [-nyos/2 ... nyos/2-1]
                    float dx = sqrtf((kx-X)*(kx-X) + (ky-Y)*(ky-Y));
                    float wgt = gridkernel(dx, kernwidth, gridos);
                    //float wgt = gridkernel(kx-X, kernwidth, gridos) * gridkernel(ky-Y, kernwidth, gridos);
                    int ridx = (r * nro) / nxos;
                    for (int ch = 0; ch < nchan && wgt > 0.f; ch++) {
                        utmp[ch] += wgt*nudata[nchan*(nro*pe + ridx + nro/2) + ch];
                        //utmp[ch].x = __fmaf_rn(wgt, nudata[nchan*(nro*pe + ridx + nro/2) + ch].x, utmp[ch].x);
                        //utmp[ch].y = __fmaf_rn(wgt, nudata[nchan*(nro*pe + ridx + nro/2) + ch].y, utmp[ch].y);
                    }
                }
            }
        }
        for (int ch = 0; ch < nchan; ++ch)
            udata[nchan*id + ch] = utmp[ch] / nxos / sqrtf(gridos);
    }
}


/*  generate 2D radial data from an input 2D image */
__global__ void
degridradial2d (
    /* udata: [NCHAN x NGRID x NGRID], nudata: NCHAN x NRO x NPE */
    float2 *nudata, const float2 * __restrict__ udata, const int n, const int nrep,
    const int nro, const int npe, const float W, const float gridos, const int skip_angles,
    const int flag_golden_angle)
{
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < nro*npe; id += blockDim.x * gridDim.x)
    {
        for (int c = 0; c < nrep; ++c) // zero my assigned unequal point
            nudata[nrep*id + c] = make_float2(0.f, 0.f);
        int pe = id / nro; // my row and column in the non-uniform data
        int ro = id % nro;
        // thread's polar coordinates
        // TODO: is this R correct?
        float R = float(ro)/float(nro) - 0.5f; // [-1/2,1/2)
        float T = flag_golden_angle ? modang(PHI*(pe + skip_angles)) : pe*M_PI/float(npe);

        // thread's Cartesian coordinates
        float X = n*R*sinf(T) + (n + 1)/2; // TODO: use _sincosf?
        float Y = n*R*cosf(T) + (n + 1)/2; // [0, n)
        for (int xu = ceilf(X-W); xu <= (X+W); ++xu)
         {
            //float wgtx = gridkernel(xu-X, W, gridos) / n;
            for (int yu = ceilf(Y-W); yu <= (Y+W); ++yu)
            {   // loop through contributing Cartesian points
                float dx = sqrtf((xu-X)*(xu-X) + (yu-Y)*(yu-Y));
                float wgt = gridkernel(dx, W, gridos) / sqrtf(npe*nro);
                //float wgt = wgtx * gridkernel(yu-Y, W, gridos);
                int i = (xu + n) % n; // periodic domain
                int j = (yu + n) % n;
                int offset = nrep*(i*n + j);
                for (int c = 0; c < nrep; ++c) {  // TODO: use nutmp temp array here for better cache usage
                    nudata[nrep*id + c] += wgt*udata[offset + c];
                    //utmp[ch].x = __fmaf_rn(wgt,nudata[nchan*(nro*pe + ridx + nro/2) + ch].x, utmp[ch].x);
                    //nudata[nrep*id + c].x = __fmaf_rn(wgt, nudata[nrep*id + c].x, udata[offset + c].x);
                    //nudata[nrep*id + c].y = __fmaf_rn(wgt, nudata[nrep*id + c].y, udata[offset + c].y);
                }
            }

        }
    }
}

void
tron_init ()
{
    if (MULTI_GPU) {
        cuTry(hipGetDeviceCount(&ndevices));
    } else
        ndevices = 1;
    DPRINT("MULTI_GPU = %d\n", MULTI_GPU);
    DPRINT("NSTREAMS = %d\n", NSTREAMS);
    DPRINT("Using %d CUDA devices\n", ndevices);
    DPRINT("Kernels configured with %d blocks of %d threads\n", threads, blocks);

    d_datasize = nc*nt*max(nro*npe1work, nxos*nyos)*sizeof(float2);  // input data

    for (int j = 0; j < NSTREAMS; ++j) // allocate data and initialize apodization and kernel texture
    {
        DPRINT("init STREAM %d\n", j);
        if (MULTI_GPU)
            hipSetDevice(j % ndevices);
        cuTry(hipStreamCreate(&stream[j]));
        fft_init(&fft_plan[j], nx, ny, nc);
        cufftSafeCall(hipfftSetStream(fft_plan[j], stream[j]));
        fft_init(&fft_plan_os[j], nxos, nyos, nc);
        cufftSafeCall(hipfftSetStream(fft_plan_os[j], stream[j]));
        cuTry(hipMalloc((void **)&d_u[j], d_datasize));
        cuTry(hipMalloc((void **)&d_v[j], d_datasize));
    }
}

void
tron_shutdown()
{
    DPRINT("freeing device memory ... ");
    for (int j = 0; j < NSTREAMS; ++j) { // free allocated memory
        if (MULTI_GPU)
            hipSetDevice(j % ndevices);
        cuTry(hipFree(d_u[j]));
        cuTry(hipFree(d_v[j]));
        hipStreamDestroy(stream[j]);
    }
    DPRINT("done.\n");
}


void
tron_nufft_adj_radial2d (float2 *d_out, float2 *d_in, const int j)
{
    // NUFFT adjoint begin
    precompensate<<<threads,blocks,0,stream[j]>>>(d_in, nc*nt, nro, npe1work);
    gridradial2d<<<threads,blocks,0,stream[j]>>>(d_out, d_in, nxos, nc*nt, nro, npe1work, kernwidth,
        gridos, skip_angles+peoffset, flags.golden_angle);
    fftshift<<<threads,blocks,0,stream[j]>>>(d_in, d_out, nxos, nt*nc, FFT_SHIFT_INVERSE);
    cufftSafeCall(hipfftExecC2C(fft_plan_os[j], d_in, d_out, HIPFFT_BACKWARD));
    fftshift<<<threads,blocks,0,stream[j]>>>(d_in, d_out, nxos, nc*nt, FFT_SHIFT_FORWARD);
    crop<<<threads,blocks,0,stream[j]>>>(d_out, nx, ny, d_in, nxos, nyos, nc*nt);
    deapodkernel<<<threads,blocks,0,stream[j]>>>(d_out, nx, nc*nt, kernwidth, gridos);
}

void
tron_nufft_radial2d (float2 *d_out, float2 *d_in, const int j)
{
    deapodkernel<<<threads,blocks,0,stream[j]>>>(d_in, nx, nc*nt, kernwidth, gridos);
    fftshift<<<threads,blocks,0,stream[j]>>>(d_out, d_in, nx, nc*nt, FFT_SHIFT_FORWARD);
    cufftSafeCall(hipfftExecC2C(fft_plan[j], d_out, d_out, HIPFFT_FORWARD));
    fftshift<<<threads,blocks,0,stream[j]>>>(d_in, d_out, nx, nc*nt, FFT_SHIFT_INVERSE);
    degridradial2d<<<threads,blocks,0,stream[j]>>>(d_out, d_in, nx, nc*nt,
         nro, npe1work, kernwidth, gridos, skip_angles, flags.golden_angle);
}

void
copy (float2* d_dst, float2* d_src, const size_t N, const int j)
{
  cuTry(hipMemcpyAsync(d_dst, d_src, N*sizeof(float2), hipMemcpyDeviceToDevice, stream[j]));
}


__global__ void
Caxpy (float2 *d_z, float2 *d_y, float2 *d_x, float alpha, const size_t N)
{
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < N; id += blockDim.x * gridDim.x)
        d_z[id] = d_y[id] +  alpha*d_x[id];
}

void
tron_cgnr_radial2d (float2* d_out, float2 *d_in, const int j, const int niter)
{
    // Based on Algorithm 1 from Knopp et al. 2007, Intl J of Biomed Imag
    // TODO: split into one pointer per stream?
    float2 *d_ztilde, *d_p, *d_ptilde, *d_r;
    float alpha, beta, res1, res2;
    float2 zres;
    const int inc = 1;
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);


    const size_t N = nx*ny*nc*nt;
    const size_t n = nro*npe1*nc*nt;
    dprint(N,d);
    dprint(n,d);
    cuTry(hipMalloc((void **)&d_ztilde, d_datasize));
    cuTry(hipMalloc((void **)&d_p, d_datasize));
    cuTry(hipMemset((float*)d_p, 0.f, N*2));
    cuTry(hipMalloc((void **)&d_ptilde, d_datasize));
    cuTry(hipMalloc((void **)&d_r, d_datasize));
    tron_nufft_adj_radial2d(d_ztilde, d_in, j); // ztilde = A^H W r
    copy(d_ptilde, d_ztilde, N, j); // ptilde = ztilde
    for (int t = 0; t < niter; ++t)
    {
        copy(d_u[j], d_ptilde, N, j);
        tron_nufft_radial2d(d_v[j], d_u[j], j); // v = A*ptilde
        copy(d_u[j], d_v[j], n, j);
        precompensate<<<threads,blocks,0,stream[j]>>>(d_u[j], nc*nt, nro, npe1work); // W*v
        hipblasScnrm2(handle, N, (hipComplex*)d_ztilde, inc, &res1);
        hipblasCdotc(handle, n, (hipComplex*)d_v[j], inc, (hipComplex*)d_u[j], inc, &zres);
        alpha = res1 / zres.x;
        //alpha = norm(d_ztilde, N) / dot(d_v[j], d_u[j], n);
        dprint(alpha,f);
        Caxpy<<<threads,blocks,0,stream[j]>>>(d_p, d_p, d_ptilde, alpha, N);
        Caxpy<<<threads,blocks,0,stream[j]>>>(d_r, d_r, d_v[j],  -alpha, n);

        hipblasScnrm2(handle, N, (hipComplex*)d_ztilde, inc, &res1);
        copy(d_u[j], d_r, n, j); // ptilde = ztilde
        tron_nufft_adj_radial2d(d_ztilde, d_u[j], j); // ztilde = A^H W r
        hipblasScnrm2(handle, N, (hipComplex*)d_ztilde, inc, &res2);
        beta = res2 / res1;
        dprint(beta,f);
        Caxpy<<<threads,blocks,0,stream[j]>>>(d_ptilde, d_ztilde, d_ptilde, beta, N);
    }

    copy(d_out, d_p, N, j);

    hipFree(d_p);
    hipFree(d_ptilde);
    hipFree(d_r);
    hipFree(d_ztilde);
    hipblasDestroy(handle);
}


// void tron_set_grid_oversampling (const float g) { gridos = g; }
// void tron_set_data_undersampling (const float u) { data_undersamp = u; }
// void tron_set_golden_angle () { flags.golden_angle = 1; }
// void tron_set_profile_slide (const int s ) { prof_slide = s; }
// void tron_set_skip_angles (const int s) { skip_angles = s; }

/*  Reconstruct images from 2D radial data.  This host routine calls the appropriate
    CUDA kernels in the correct order depending on the direction of recon.   */

__host__ void
recon_radial2d (float2 *h_outdata, const float2 *__restrict__ h_indata)
{
    DPRINT("recon_radial2d\n");
    tron_init();

    for (int z = 0; z < nz; ++z)
    {
        int j = z % NSTREAMS; // j is the stream index
        if (MULTI_GPU) hipSetDevice(j % ndevices);

        peoffset = z*prof_slide;
        size_t data_offset = nc*nt*nro*peoffset;  // address offsets into the data arrays
        size_t img_offset = nt*nx*ny*z;

        printf("[dev %d, stream %d] reconstructing slice %d/%d from PEs %d-%d\n",
            j%ndevices, j, z+1, nz, z*prof_slide, (z+1)*prof_slide-1);

        if (flags.adjoint) { // copy working data to GPU
            cuTry(hipMemcpyAsync(d_u[j], h_indata + data_offset,
                nc*nt*nro*npe1work*sizeof(float2), hipMemcpyHostToDevice, stream[j]));
        } else {
            cuTry(hipMemcpyAsync(d_u[j], h_indata + data_offset,
                nc*nt*nx*ny*sizeof(float2), hipMemcpyHostToDevice, stream[j]));
        }
        int niter = 0;
        if (flags.adjoint) {  // process data resident on GPU
            for (int t = 0; t < niter; ++t) {
              tron_nufft_adj_radial2d(d_v[j], d_u[j], j);
              tron_nufft_radial2d(d_u[j], d_v[j], j);
            }
            tron_nufft_adj_radial2d(d_v[j], d_u[j], j);

            //tron_cgnr_radial2d (d_v[j], d_u[j], j, niter);
        } else
            tron_nufft_radial2d(d_v[j], d_u[j], j);


        if (flags.adjoint)  // send result back to CPU
        {
            coilcombinesos<<<threads,blocks,0,stream[j]>>>(d_u[j], d_v[j], nx, nc); // TODO: should this have nt in it?
            // TODO: look at nc to decide whether to coil combine and by how much (can compress)
            //coilcombinewalsh<<<threads,blocks,0,stream[j]>>>(d_img[j],d_coilimg[j], nx, nc, nt, 1); /* 0 works, 1 good, 3 better */
#ifdef CUDA_HOST_MALLOC
            cuTry(hipMemcpyAsync(h_outdata + img_offset, d_u[j],
                nx*ny*nt*sizeof(float2), hipMemcpyDeviceToHost, stream[j]));
#else
            cuTry(hipMemcpy(h_outdata + img_offset, d_u[j],
                nx*ny*nt*sizeof(float2), hipMemcpyDeviceToHost));
#endif
        } else {
#ifdef CUDA_HOST_MALLOC
            cuTry(hipMemcpyAsync(h_outdata + nc*nt*nro*npe1work*z, d_v[j],
                nc*nt*nro*npe1work*sizeof(float2), hipMemcpyDeviceToHost, stream[j]));
#else
            cuTry(hipMemcpyAsync(h_outdata + nc*nt*nro*npe1work*z, d_v[j],
                nc*nt*nro*npe1work*sizeof(float2), hipMemcpyDeviceToHost));
#endif
        }
    }

    tron_shutdown();
}




}

void
print_usage()
{
    fprintf(stderr, "Trajectory-optimized Non-uniform Fast Fourier Transform\n");
    fprintf(stderr, "Usage: tron [-3ahuv] [-r cmds] [-d prof_slide] [-k width] [-o gridos] [-s skip_angles] [-u data_undersamp] <infile.ra> [outfile.ra]\n");
    fprintf(stderr, "\t-3\t\t\t3D koosh ball trajectory\n");
    fprintf(stderr, "\t-a\t\t\tadjoint operation\n");
    fprintf(stderr, "\t-d prof_slide\t\tnumber of phase encodes to slide between slices for helical scans\n");
    fprintf(stderr, "\t-g\t\t\tgolden angle radial\n");
    fprintf(stderr, "\t-h\t\t\tshow this help\n");
    fprintf(stderr, "\t-k width\t\twidth of gridding kernel\n");
    fprintf(stderr, "\t-o gridos\t\tgrid oversampling factor\n");
    fprintf(stderr, "\t-r nro\t\t\tnumber of readout points\n");
    fprintf(stderr, "\t-s skip_angles\t\tnumber of initial phase encodes to skip\n");
    fprintf(stderr, "\t-u data_undersamp\tinput data undersampling factor\n");
    fprintf(stderr, "\t-v\t\t\tverbose output\n");
}


int
main (int argc, char *argv[])
{
    float2 *h_indata, *h_outdata;
    ra_t ra_in, ra_out;
    int c, index;
    char infile[1024], outfile[1024];

    opterr = 0;
    while ((c = getopt (argc, argv, "3ad:ghk:o:r:s:u:v")) != -1)
    {
        switch (c) {
            case '3':
                flags.koosh = 1;
            case 'a':
                flags.adjoint = 1;
                break;
            case 'd':
                prof_slide = atoi(optarg);
                break;
            case 'g':
                flags.golden_angle = 1;
                break;
            case 'h':
                print_usage();
                return 1;
            case 'k':
                kernwidth = atof(optarg);
                break;
            case 'o':
                gridos = atof(optarg);
                break;
            case 'u':
                data_undersamp = atof(optarg);
                break;
            case 'r':
                nro = atoi(optarg);
                break;
            case 's':
                skip_angles = atoi(optarg);
                break;
            case 'v':
                flags.verbose = 1;
                break;
            default:
                print_usage();
                return 1;
        }
    }

    // set input and output files
    snprintf(outfile, 1024, "img_tron.ra"); // default value
    if (argc == optind) {
       print_usage();
       return 1;
    }
    for (index = optind; index < argc; index++) {
      if (index == optind)
        snprintf(infile, 1024, "%s", argv[index]);
      else if (index == optind + 1)
        snprintf(outfile, 1024, "%s", argv[index]);
    }

    dprint(skip_angles, d);
    dprint(prof_slide, d);
    dprint(kernwidth, .1f);
    dprint(data_undersamp, .3f);
    dprint(gridos, .3f);
    dprint(infile, s)
    dprint(outfile, s)

    DPRINT("Reading %s\n", infile);
    ra_read(&ra_in, infile);
    h_indata = (float2*)ra_in.data;
    assert(ra_in.ndims == 5);
    DPRINT("Sanity check: indata[0] = %f + %f i\n", h_indata[0].x, h_indata[0].y);
    DPRINT("indims = {%llu, %llu, %llu, %llu, %llu}\n", ra_in.dims[0], ra_in.dims[1], ra_in.dims[2], ra_in.dims[3], ra_in.dims[4]);
    DPRINT("WARNING: Assuming square Cartesian dimensions for now.\n");

    ra_out.ndims = 5;
    ra_out.dims = (uint64_t*) malloc(ra_out.ndims*sizeof(uint64_t));
    ra_out.dims[0] = 1;
    ra_out.flags = 0;
    ra_out.eltype = 4;
    ra_out.elbyte = 8;

    // HERE IS WHERE WE COMPUTE OUTPUT DIMENSIONS BASED ON INPUT AND OPTIONAL ARGS
    if (flags.adjoint)
    {
        nc = ra_in.dims[0];
        nt = ra_in.dims[1];
        nro = ra_in.dims[2];
        npe1 = ra_in.dims[3];
        npe2 = ra_in.dims[4];
        nx = nro / 2;
        ny = nro / 2;
        nxos = nx * gridos;
        nyos = ny * gridos;
        if (npe1 <= nro * data_undersamp)  /* must be implicitly undersampled */
            npe1work = npe1;
        else
            npe1work = nro * data_undersamp;
        if (prof_slide == 0)
            prof_slide = npe1work;
        if (flags.koosh) {
            nz = nro / 2;
            nzos = nz * gridos;
        } else {
            nz = 1 + (npe1 - npe1work) / prof_slide;
            nzos = 1;
        }
        //npe2work = npe2;
        ra_out.dims[1] = nt;
        ra_out.dims[2] = nx;
        ra_out.dims[3] = ny;
        ra_out.dims[4] = nz;
        h_outdatasize = 1*nt*nx*ny*nz*sizeof(float2);
    }
    else
    {
        nc = ra_in.dims[0];
        nt = ra_in.dims[1];
        nx = ra_in.dims[2];
        ny = ra_in.dims[3];
        nz = ra_in.dims[4];
        nxos = nx*gridos;
        nyos = ny*gridos;
        nro = gridos*nx;  // TODO: implement non-square images
        npe1work = data_undersamp * nro;
        npe1 = npe1work;
        if (flags.koosh) {
            npe2 = nz;
            nzos = nz; //gridos*nz ?;
        } else {
            npe2 = 1;
            nzos = 1;
        }
        ra_out.dims[1] = nt;
        ra_out.dims[2] = nro;
        ra_out.dims[3] = npe1;
        ra_out.dims[4] = npe2;
        gridos = 1.f;
        h_outdatasize = nc*nt*nro*npe1*npe2*sizeof(float2);
    }
    ra_out.size = h_outdatasize;
    dprint(h_outdatasize,ld);
    assert(nc % 2 == 0 || nc == 1); // only single or even dimensions implemented for now

    dprint(data_undersamp,f);
    dprint(gridos,f);
    dprint(nc,d);
    dprint(nt,d);
    dprint(nro,d);
    dprint(npe1,d);
    dprint(npe2,d);
    dprint(nx,d);
    dprint(ny,d);
    dprint(nz,d);
    dprint(nxos,d);
    dprint(nyos,d);
    dprint(nzos,d);
    dprint(npe1work,d);


#ifdef CUDA_HOST_MALLOC
    // allocate pinned memory, which allows async calls
    cuTry(hipHostMalloc((void**)&h_outdata, h_outdatasize));
#else
    h_outdata = (float2*)malloc(h_outdatasize);
#endif

    DPRINT("Running reconstruction ...\n ");
    clock_t start = clock();

    recon_radial2d(h_outdata, h_indata);

    clock_t end = clock();
    DPRINT("Elapsed time: %.2f s\n", ((float)(end - start)) / CLOCKS_PER_SEC);

    DPRINT("Saving result to %s\n", outfile);
    ra_out.data = (uint8_t*)h_outdata;
    ra_write(&ra_out, outfile);


    DPRINT("Cleaning up.\n");
    ra_free(&ra_in);
#ifdef CUDA_HOST_MALLOC
    hipHostFree(&h_outdata);
#else
    free(h_outdata);
#endif
    hipDeviceReset();

    return 0;
}
